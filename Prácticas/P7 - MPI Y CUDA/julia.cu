#include "hip/hip_runtime.h"
/*
	Autor: Jose Antonio Padial Molina
	Curso: 2017/2018
	Asignatura: ACAP
	Profesor: María Isabel García Arenas
*/

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#define TX 512
#define TY 512

__device__ short CalcJulia(int x_ind, int y_ind){

	float cr=0.626;		// Valor de c
	float ci=0.0;
	float oror = -1.0;
	float oi = -1.0;
	float dr=1,di=1;		// Esq superior derecha del espacio a evaluar (encuadre)

	float tamr=dr-oror;
	float tami=di-oi;
	float incr=tamr/(float)TX;
	float inci=tami/(float)TY;

	float zr,zi;
	float zrs,zis;
	short color;

	zi=(float)oi+(float)y_ind*incr;
	zr=(float)oror+(float)x_ind*inci;
	zrs=zis=(float)0; // semilla Z0
	color=0;	    // colorea y limita las iteraciones de puntos del conjunto

	while (zrs + zis < (float) 4 && color < 256) {
		zrs = zr * zr;
	zis = zi * zi;
	float zrsaux = zr;

		zr=(-(float)7*zr*pow(zi,6)) + ((float)35*pow(zr,3)*pow(zi,4)) - ((float)21*pow(zr,5)*pow(zi,2))+pow(zr,7) + cr;
		zi=-pow(zi,7)+((float)21*pow(zi,5)*pow(zrsaux,2))-((float)35*pow(zrsaux,4)*pow(zi,3)) + ((float)7*pow(zrsaux,6)*zi)  + ci;
		color++;
}

	return color;
}

__global__ void JuliaKernel(char *imagen){
	int i = threadIdx.x + blockIdx.x*blockDim.x; //
	int j = blockIdx.y;

	if (i < TX && j < TY) imagen[i + j*gridDim.x*blockDim.x] = CalcJulia(i, j); // Escribo en la columna i y fila j
}

int main(int argc, char* argv[]) {

	int i;

	struct timeval start, end;
	float gpu_time; // GPU time wm(with memory)

	gettimeofday(&start, NULL);

	char *im = (char*) malloc(TX*TY*sizeof(char));
	char *im_d; //Imagen en la memoria de la GPU

	hipMalloc((void **) &im_d, TX*TY*sizeof(char));


	dim3 DimGrid(TX/512,TY,1); // 512 is the maximum number of threads per block.
	dim3 DimBlock(512,1,1); //Number of threads per block

	JuliaKernel<<<DimGrid,DimBlock>>>(im_d);
	hipDeviceSynchronize();

	hipMemcpy(im, im_d, TX*TY*sizeof(char), hipMemcpyDeviceToHost);

	gettimeofday(&end, NULL);

	gpu_time = (end.tv_sec - start.tv_sec) * 1000.0;      // sec to ms
     	gpu_time += (end.tv_usec - start.tv_usec) / 1000.0;   // us to ms

	printf("%d\t%f\n", TX, gpu_time);

	 FILE *fim;
	 fim=fopen("julia.raw","wb");
	 if (!fim)
	 {
	     printf("No se puede abrir el fichero de salida.\n");
	     exit(1);
	 }
	 for(i=0;i < TY;++i){
	 	fwrite(&(im[i*TX]),sizeof(char),TX,fim);
	 }
	 fclose(fim);

	hipFree(im_d);
	free(im);

	return EXIT_SUCCESS;
}
