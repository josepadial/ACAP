/*
    @autor Jose Antonio Padial Molina
    Archivo que muestra las caracteristicas de nuestros Devices
*/
#include "hip/hip_runtime.h"
#include ""
#include<stdio.h>
#include<stdlib.h>
#include<time.h>
int main() {
  int numeroDevice;
  hipGetDeviceCount(&numeroDevice);
  for (int i = 0; i < numeroDevice; i++) {
    hipDeviceProp_t propiedades;
    hipGetDeviceProperties(&propiedades, i);
    printf("Dispositivo Numero: %d\n", i);
    printf("  Nombre del dispositivo: %s\n", propiedades.name);
    printf("  Frecuencia Reloj (KHz): %d\n",
           propiedades.memoryClockRate);
    printf("  Memory Bus Width (bits): %d\n",
           propiedades.memoryBusWidth);
    printf("  Ancho de Banda (GB/s): %f\n\n",
           2.0*propiedades.memoryClockRate*(propiedades.memoryBusWidth/8)/1.0e6);
  }
}
