#include "hip/hip_runtime.h"
/*
	Autor: Jose Antonio Padial Molina
	Curso: 2017/2018
	Asignatura: ACAP
	Profesor: María Isabel García Arenas
*/

#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
#include <time.h>

using namespace std;

//Función para redondear la C prima
//Utilizaremos la función ceil, que se encarga de redondear el argumento redondeado hacia arriba
void redondeo(float &numero0, float &numero1, double cifras){

	double i, j;

	i=((numero0*pow(10,cifras))-0.02)/pow(10,cifras);
	j=((numero1*pow(10,cifras))-0.02)/pow(10,cifras);

	numero0=ceil(i);
	numero1=ceil(j);
}


//Funcion de "nucleo" que calcula la suma dando a cada hebra la
//funcion de calcular la suma de los elementos de una posicion
__global__ void vecAddKernel(float *A, float *B, float *C, int n){

	int i = threadIdx.x+blockDim.x*blockIdx.x;
	if(i<n)
	  C[i] = A[i]+B[i];
}




int leeDimension (char fichero [20]){
	fstream filein1;

	filein1.open(fichero, ios::in);
		if(!filein1){
			cerr<<"No se pudo abrir el fichero1"<<endl;
			exit(1);
		}
	int filas=0;
	char caracter;
	while( filein1.get(caracter) ){
		if(caracter=='\n')
			filas++;
	}
	return filas;
}

//Función que suma en CUDA dos vectores
void vecAdd(float *hA, float *hB, float *hC, int n,  double &ncgt){
	int size = n*sizeof(float);
	float * dA, *dB, *dC;
	struct timespec cgt1,cgt2;
	int BlockDim;

	//Reservamos memoria y movemos las entradas a la memoria del device
	hipMalloc((void **) &dA,size);
	hipMemcpy(dA,hA,size,hipMemcpyHostToDevice);
	hipMalloc((void **) &dB,size);
	hipMemcpy(dB,hB,size,hipMemcpyHostToDevice);
	hipMalloc((void **) &dC,size);

	if(n>1024)
		BlockDim=512;
	else
		BlockDim=ceil(n/2)+1;

	dim3 dimblock(BlockDim,1,1);
	dim3 dimgrid(ceil(n/BlockDim)+1 ,1,1);

//Iniciamos el nucleo para calcular la suma
	clock_gettime(CLOCK_REALTIME,&cgt1);
	vecAddKernel<<< dimgrid, dimblock>>>(dA, dB, dC, n);
 	clock_gettime(CLOCK_REALTIME,&cgt2);

	ncgt=(double) (cgt2.tv_sec-cgt1.tv_sec)+
	(double) ((cgt2.tv_nsec-cgt1.tv_nsec)/(1.e+9));

//Copiamos el resultado del host
	hipMemcpy(hC,dC,size,hipMemcpyDeviceToHost);
//Liberamos memoria
	hipFree(dA);hipFree(dB);hipFree(dC);
}





int main(int argc, char **argv){

	if(argc< 4){
		cerr<<"Error al introducir argumentos, el orden de los vectores sería:\nVector1\nVector2\nOutput\n";
		exit(1);
	}
	printf("********************TARJETA***************************\n" );
	int numeroDevice;
  hipGetDeviceCount(&numeroDevice);
  for (int i = 0; i < numeroDevice; i++) {
    hipDeviceProp_t propiedades;
    hipGetDeviceProperties(&propiedades, i);
    printf("Dispositivo Numero: %d\n", i);
    printf("  Nombre del dispositivo: %s\n", propiedades.name);
    printf("  Frecuencia Reloj (KHz): %d\n",
           propiedades.memoryClockRate);
    printf("  Memory Bus Width (bits): %d\n",
           propiedades.memoryBusWidth);
    printf("  Ancho de Banda (GB/s): %f\n\n",
           2.0*propiedades.memoryClockRate*(propiedades.memoryBusWidth/8)/1.0e6);
  }

	printf("********************TARJETA***************************\n" );
	int n=leeDimension(argv[1]);

	// cout<< endl<<" Tamanno Vectores = "<<n<<endl;

	float *h_A, *h_B, *h_C, *vector_output;

	h_A= new float[n];
	h_B=new float[n];
	h_C=new float[n];
	vector_output=new float[n];

//Leemos el fichero
	ifstream imput0 (argv[1], ifstream::in);
	ifstream imput1 (argv[2], ifstream::in);
	ifstream output (argv[3], ifstream::in);

	char c[20];
//Convertimos la cadena a un float
	for(int i=0; i<=n; i++){
		imput0 >>c;
		h_A[i]=atof(c);

	 	imput1 >>c;
		h_B[i]=atof(c);

		output >>c;
		vector_output[i]=atof(c);
	}

// almacenamiento de n elementos h_A,h_B y h_C

	double ncgt;
	vecAdd(h_A, h_B, h_C, n, ncgt);



//Mostrar vectores y resultados
 /*
	for(int i=0; i<filas; i++)
		cout<<"h_C["<<i<<"]="<<h_C[i]<<"	 	vector_output["<<i<<"]="<<vector_output[i]<<endl;
*/
for(int i= 1;i<=n;i++)
cout<<h_A[i]<<"+"<<h_B[i]<<"="<<h_C[i]<<endl;

// Comprobacion
	int linea=0;
	bool no_falla=true;

	for(int i=1; i<n && no_falla; i++){
		redondeo(h_C[i],vector_output[i],2);
		if(h_C[i]!=vector_output[i]){
			no_falla=false;
			linea=i;
		}
	}

	if(no_falla)
		cout<<"\nResultados correctos"<<endl;


	cout.precision(8);
	cout.setf(ios::fixed);


	cout<<"Tamaño del vector : "<<n<<"\nTiempo de ejecución : "<<ncgt<<endl<<endl;

}
